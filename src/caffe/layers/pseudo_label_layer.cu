#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layers/pseudo_label_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Get_roi_det_blob(const int nthreads, const Dtype* roi_data,
                                 const Dtype* det_data, const int num_roi,
                                 const int num_det, Dtype* roi_det_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int roi_idx = index / num_det;
    const int det_idx = index % num_det;

    // For each ROI R = [batch_index x1 y1 x2 y2]: max pool over R
    const Dtype* roi = roi_data + roi_idx * 5;
    //[image_id, label, confidence, xmin, ymin, xmax, ymax]
    const Dtype* det = det_data + det_idx * 7;

    if (roi[0] != det[0]) {
      roi_det_data[roi_idx * num_det + roi_idx] = 0;
      continue;
    }

    roi += 1;
    det += 3;
    if (roi[0] > det[2] || roi[2] < det[0] || roi[1] > det[3] ||
        roi[3] < det[1]) {
      roi_det_data[roi_idx * num_det + roi_idx] = 0;
      continue;
    }

    const Dtype intersect_x1 = max(roi[0], det[0]);
    const Dtype intersect_y1 = max(roi[1], det[1]);
    const Dtype intersect_x2 = min(roi[2], det[2]);
    const Dtype intersect_y2 = min(roi[3], det[3]);

    const Dtype intersect_h = intersect_y2 - intersect_y1;
    const Dtype intersect_w = intersect_x2 - intersect_x1;
    if (intersect_h == 0 && intersect_w == 0) {
      roi_det_data[roi_idx * num_det + roi_idx] = 0;
      continue;
    }
    const Dtype intersect_size = intersect_h * intersect_w;
    const Dtype roi_size = (roi[2] - roi[0]) * (roi[3] - roi[1]);
    const Dtype det_size = (det[2] - det[0]) * (det[3] - det[1]);

    roi_det_data[roi_idx * num_det + roi_idx] =
        intersect_size / (roi_size + det_size - intersect_size);
  }
}

template <typename Dtype>
void PseudoLabelLayer<Dtype>::top1forward(const vector<Blob<Dtype>*>& bottom,
                                          const vector<Blob<Dtype>*>& top) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  Get_roi_det_blob<
      Dtype><<<CAFFE_GET_BLOCKS(num_roi_ * num_det_), CAFFE_CUDA_NUM_THREADS>>>(
      num_roi_ * num_det_, bottom[1]->gpu_data(), bottom[3]->gpu_data(),
      num_roi_, num_det_, roi_det_.mutable_gpu_data());

  caffe_gpu_set(det_cls_.count(), Dtype(0.), det_cls_.mutable_gpu_data());
  Dtype* det_cls_data = det_cls_.mutable_cpu_data();
  //[image_id, label, confidence, xmin, ymin, xmax, ymax]
  const Dtype* det_data = bottom[3]->cpu_data();
  for (int i = 0; i < num_det_; ++i) {
    const Dtype* det = det_data + i * 7;
    const int label = det[1];
    CHECK_LT(label, 0) << "found background label in detection result.";
    det_cls_data[i * num_cls_ + label - 1] = det[2];
  }

  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_roi_, num_cls_,
                        num_det_, Dtype(1.), roi_det_.gpu_data(),
                        det_cls_.gpu_data(), Dtype(0.),
                        top[1]->mutable_gpu_data());
}

template <typename Dtype>
void PseudoLabelLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                          const vector<Blob<Dtype>*>& top) {
  top0forward(bottom, top);
  if (bottom.size() == 4) {
    top1forward(bottom, top);
  }
}

template <typename Dtype>
void PseudoLabelLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                           const vector<bool>& propagate_down,
                                           const vector<Blob<Dtype>*>& bottom) {
  for (size_t i = 0; i < bottom.size(); i++) {
    if (propagate_down[i]) {
      caffe_set(bottom[i]->count(), static_cast<Dtype>(0),
                bottom[i]->mutable_cpu_diff());
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(PseudoLabelLayer);

}  // namespace caffe
